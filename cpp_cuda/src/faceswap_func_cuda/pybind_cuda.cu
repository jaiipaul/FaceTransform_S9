#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include <stdlib.h>
#include "pybind_cuda.cuh"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

__global__ void gpu_square(int* v, int size){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ){
        v[id] = (255-v[id]);
    }
}


__global__ void kernel_ApplyAllHomography(int width, int height, 
                                          int* imgLabel, double* H_flat, 
                                          double* XI,    double* YI){
    int i = blockIdx.x*blockDim.x + threadIdx.x,
		j = blockIdx.y*blockDim.y + threadIdx.y;
    double den;
    int label;
    if (i < width  && j < height){
        //printf("%d | %d\n", i, j);
        label = imgLabel[j*width+i];
        if (label > 0){
            label -= 1;
            den = 1.0 / (H_flat[9*label + 2]*i + H_flat[9*label + 5]*j + 1);
            
            XI[j*width + i] = (H_flat[9*label    ]*i+H_flat[9*label + 3]*j+H_flat[9*label + 6])*den;
            YI[j*width + i] = (H_flat[9*label + 1]*i+H_flat[9*label + 4]*j+H_flat[9*label + 7])*den;
        }
    }
}   

__global__ void kernel_RecreateImage(int* imgOut, 
                                     int* imgCAM ,int width_CAM, int height_CAM, 
                                     int* imgFTA, int width_FTA, int height_FTA, 
                                     double* XI, double* YI, int* imgLabel){
    int x_CAM = blockIdx.x*blockDim.x + threadIdx.x,
		y_CAM = blockIdx.y*blockDim.y + threadIdx.y;

    int x_FTA, y_FTA;

    if (x_CAM < width_CAM  && y_CAM < height_CAM){
        if(imgLabel[y_CAM*width_CAM+x_CAM] > 0){
            x_FTA = (int)XI[y_CAM * width_CAM + x_CAM];
            y_FTA = (int)YI[y_CAM * width_CAM + x_CAM];  
  
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 0] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 0];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 1] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 1];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 2] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 2];          
        }
    }          
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = BLOCK_SIZE_X*BLOCK_SIZE_Y;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    //printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}

void ApplyAllHomography_CUDA(int width, int height, int* imgLabel, double* H_flat, double* XI, double* YI, int n_quadrangles)
{   
    int size = width*height;
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);
    dim3 dimGrid((width+nbThreadx-1)/nbThreadx, (height+nbThready-1)/nbThready);

    /* Copy Homography matrix */
    double *H_in_cuda = NULL;
    hipMalloc((void **)&H_in_cuda, 9*n_quadrangles*sizeof(double));
    hipMemcpy(H_in_cuda, H_flat, 9*n_quadrangles*sizeof(double), hipMemcpyHostToDevice);

    /* Copy Label Image */
    int * ImgLabel_in_cuda = NULL;
    hipMalloc((void **)&ImgLabel_in_cuda, size*sizeof(int));
    hipMemcpy(ImgLabel_in_cuda, imgLabel, size*sizeof(int), hipMemcpyHostToDevice);

    /* Allocate memory for interpolation matrix */
    double *XI_out_cuda = NULL;
    double *YI_out_cuda = NULL;
    hipMalloc((void **)&XI_out_cuda, size*sizeof(double));
    hipMalloc((void **)&YI_out_cuda, size*sizeof(double));
    
    kernel_ApplyAllHomography<<<dimGrid, dimBlock>>>(width, height, ImgLabel_in_cuda, H_in_cuda, XI_out_cuda, YI_out_cuda);

    hipDeviceSynchronize();
    /* Copy device result to host memory */
    hipMemcpy(XI, XI_out_cuda, size*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(YI, YI_out_cuda, size*sizeof(double), hipMemcpyDeviceToHost);

    /* Free device memory */
    hipFree(ImgLabel_in_cuda);
    hipFree(H_in_cuda);
    hipFree(XI_out_cuda);
    hipFree(YI_out_cuda);
}

void RecreateImage_CUDA(int* imgOut,
	               int* imgCAM, int width_CAM, int height_CAM,
	               int* imgFTA, int width_FTA, int height_FTA,
 	               double* XI, double* YI, int* imgLabel)
{
	int size_CAM = width_CAM * height_CAM;
    int size_FTA = width_FTA * height_FTA;

	int* imgout_cuda = NULL, *imgCAM_cuda = NULL, *imgFTA_cuda = NULL, *imgLabel_cuda = NULL;
	double *XI_cuda = NULL, *YI_cuda = NULL;
	
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);
    dim3 dimGrid((width_CAM+nbThreadx-1)/nbThreadx, (height_CAM+nbThready-1)/nbThready);

	hipMalloc((void **)&imgCAM_cuda, 3*size_CAM*sizeof(int));
	hipMemcpy(imgCAM_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&imgFTA_cuda, 3*size_FTA*sizeof(int));	
	hipMemcpy(imgFTA_cuda, imgFTA, 3*size_FTA*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&imgout_cuda, 3*size_CAM*sizeof(int));
    hipMemcpy(imgout_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&XI_cuda, size_CAM * sizeof(double));
	hipMemcpy(XI_cuda, XI, size_CAM * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&YI_cuda, size_CAM * sizeof(double));
	hipMemcpy(YI_cuda, YI, size_CAM * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&imgLabel_cuda, size_CAM * sizeof(int));
	hipMemcpy(imgLabel_cuda, imgLabel, size_CAM * sizeof(int), hipMemcpyHostToDevice);



    /* Kernel */
	kernel_RecreateImage<<<dimGrid, dimBlock>>>(imgout_cuda, 
                                                imgCAM_cuda, width_CAM, height_CAM, 
                                                imgFTA_cuda, width_FTA, height_FTA,
                                                XI_cuda, YI_cuda, imgLabel_cuda);	

	hipDeviceSynchronize();	
	
    /* Copy result to host memory */
    hipMemcpy(imgOut, imgout_cuda, 3*size_CAM*sizeof(int), hipMemcpyDeviceToHost);
	
    /* Free device memory */
    hipFree(XI_cuda);
	hipFree(YI_cuda);
	hipFree(imgCAM_cuda);
	hipFree(imgFTA_cuda);
	hipFree(imgLabel_cuda);
	hipFree(imgout_cuda);	
}