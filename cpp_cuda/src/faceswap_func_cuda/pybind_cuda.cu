#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include <stdlib.h>
#include "pybind_cuda.cuh"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

__global__ void gpu_square(int* v, int size){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ){
        v[id] = (255-v[id]);
    }
}


__global__ void kernel_ApplyAllHomography(int width, int height, int* imgLabel, double* H_flat, double* XI, double* YI){
    int i = blockIdx.x*blockDim.x + threadIdx.x,
		j = blockIdx.y*blockDim.y + threadIdx.y;
    double den;
    int label;
    if (i < width  && j < height){
        //printf("%d | %d\n", i, j);
        label = imgLabel[j*width+i];
        if (label > 0){
            den = 1.0 / (H_flat[9*label + 2]*i + H_flat[9*label + 5]*j + 1);
            
            XI[j*width + i] = (H_flat[9*label    ]*i+H_flat[9*label + 3]*j+H_flat[9*label + 6])*den;
            YI[j*width + i] = (H_flat[9*label + 1]*i+H_flat[9*label + 4]*j+H_flat[9*label + 7])*den;
        }
    }
    //__syncthreads();
}   

__global__ void kernel_RecreateImage(int* imgOut, 
                                     int* imgCAM ,int width_CAM, int height_CAM, 
                                     int* imgFTA, int width_FTA, int height_FTA, 
                                     double* XI, double* YI, int* imgLabel){
    int x_CAM = blockIdx.x*blockDim.x + threadIdx.x,
		y_CAM = blockIdx.y*blockDim.y + threadIdx.y;

    int canal;
    int x_FTA, y_FTA;

    if (x_CAM < width_CAM  && y_CAM < height_CAM){
        //printf("%d | %d\n", x_CAM, y_CAM);
        if(imgLabel[y_CAM*width_CAM+x_CAM] > 0){
            x_FTA = (int)XI[y_CAM * width_CAM + x_CAM];
            y_FTA = (int)YI[y_CAM * width_CAM + x_CAM];  
            //printf("in : %d | %d | %d \n", imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 0],
            //                               imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 1],
            //                               imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 2]);
  
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 0] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 0];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 1] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 1];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 2] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 2];          
        }
    }    
    //__syncthreads();       
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = BLOCK_SIZE_X*BLOCK_SIZE_Y;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    //printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}

void ApplyAllHomography_CUDA(int width, int height, int* imgLabel, double* H_flat, double* XI, double* YI, int n_quadrangles)
{   
    int size = width*height;
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);
    dim3 dimGrid((width+nbThreadx-1)/nbThreadx, (height+nbThready-1)/nbThready);

    double *XI_out_cuda = NULL;
    double *YI_out_cuda = NULL;

    printf("Try to copy H matrix\n");
    /* Copy Homography matrix */
    double *H_in_cuda = NULL;
    hipMalloc((void **)&H_in_cuda, 9*n_quadrangles*sizeof(double));
    hipError_t errA = hipGetLastError();
    if (errA != hipSuccess) 
        printf("Error CUDA before homography: %s\n", hipGetErrorString(errA));
    printf("hipMalloc done for H\n");
    hipMemcpy(H_in_cuda, H_flat, 9*n_quadrangles*sizeof(double), hipMemcpyHostToDevice);

    printf("Copied H matrix\n");
    /* Copy Label Image */
    int * ImgLabel_in_cuda = NULL;
    hipMalloc((void **)&ImgLabel_in_cuda, size*sizeof(int));
    hipMemcpy(ImgLabel_in_cuda, imgLabel, size*sizeof(int), hipMemcpyHostToDevice);
    printf("Copied ImgLabel\n");

    hipMalloc((void **)&XI_out_cuda, size*sizeof(double));
    hipMalloc((void **)&YI_out_cuda, size*sizeof(double));
    //hipMemcpy(XI_out_cuda, XI, size*sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(YI_out_cuda, YI, size*sizeof(double), hipMemcpyHostToDevice);
    

    printf("Kernel H\n");
    kernel_ApplyAllHomography<<<dimGrid, dimBlock>>>(width, height, ImgLabel_in_cuda, H_in_cuda, XI_out_cuda, YI_out_cuda);
    hipError_t errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA after homography: %s\n", hipGetErrorString(errB));

    //printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(XI, XI_out_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(YI, YI_out_cuda, size*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(ImgLabel_in_cuda);
    hipFree(H_in_cuda);
    hipFree(XI_out_cuda);
    hipFree(YI_out_cuda);
}

void RecreateImage_CUDA(int* imgOut,
	               int* imgCAM, int width_CAM, int height_CAM,
	               int* imgFTA, int width_FTA, int height_FTA,
 	               double* XI, double* YI, int* imgLabel)
{
	int size_CAM = width_CAM * height_CAM;
    int size_FTA = width_FTA * height_FTA;
    printf("%d\n", size_FTA);
	int* imgout_cuda = NULL, *imgCAM_cuda = NULL, *imgFTA_cuda = NULL, *imgLabel_cuda = NULL;
	double *XI_cuda = NULL, *YI_cuda = NULL;
	
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);

	hipMalloc((void **)&imgCAM_cuda, 3*size_CAM*sizeof(int));
    hipError_t errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R imgCAM: %s\n", hipGetErrorString(errB));
	hipMemcpy(imgCAM_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R imgCAM copy: %s\n", hipGetErrorString(errB));
	hipMalloc((void **)&imgFTA_cuda, 3*size_FTA*sizeof(int));	
	hipMemcpy(imgFTA_cuda, imgFTA, 3*size_FTA*sizeof(int), hipMemcpyHostToDevice);
    printf("%d\n", imgFTA[0]);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R imgFTA: %s\n", hipGetErrorString(errB));

	hipMalloc((void **)&imgout_cuda, 3*size_CAM*sizeof(int));
    hipMemcpy(imgout_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R imgOut: %s\n", hipGetErrorString(errB));

	hipMalloc((void**)&XI_cuda, size_CAM * sizeof(double));
	hipMemcpy(XI_cuda, XI, size_CAM * sizeof(double), hipMemcpyHostToDevice);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R XI: %s\n", hipGetErrorString(errB));
	hipMalloc((void**)&YI_cuda, size_CAM * sizeof(double));
	hipMemcpy(YI_cuda, YI, size_CAM * sizeof(double), hipMemcpyHostToDevice);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R YI: %s\n", hipGetErrorString(errB));

	hipMalloc((void**)&imgLabel_cuda, size_CAM * sizeof(int));
	hipMemcpy(imgLabel_cuda, imgLabel, size_CAM * sizeof(int), hipMemcpyHostToDevice);
    errB = hipGetLastError();
    if (errB != hipSuccess) 
        printf("Error CUDA R imgLabel: %s\n", hipGetErrorString(errB));	
	// appel bonne fonction
	dim3 dimGrid((width_CAM+nbThreadx-1)/nbThreadx, (height_CAM+nbThready-1)/nbThready);
    printf("Kernel R\n");
	kernel_RecreateImage<<<dimGrid, dimBlock>>>(imgout_cuda, 
                                                imgCAM_cuda, width_CAM, height_CAM, 
                                                imgFTA_cuda, width_FTA, height_FTA,
                                                XI_cuda, YI_cuda, imgLabel_cuda);	
	//
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error CUDA Recreate: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();	
	
	hipFree(XI_cuda);
	hipFree(YI_cuda);
	hipFree(imgCAM_cuda);
	hipFree(imgFTA_cuda);
	hipFree(imgLabel_cuda);
	hipMemcpy(imgOut, imgout_cuda, 3*size_CAM*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(imgout_cuda);	
}