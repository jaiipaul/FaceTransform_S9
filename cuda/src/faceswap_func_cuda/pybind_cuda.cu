#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include "pybind_cuda.cuh"

__global__ void gpu_square(int* v, int size)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ) v[id] = v[id]*v[id];
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = 512;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}

