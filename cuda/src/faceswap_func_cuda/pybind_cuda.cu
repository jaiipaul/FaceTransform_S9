#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include "pybind_cuda.cuh"

__global__ void gpu_square(int* v, int size){
    printf("in kernel\n");
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ){
        printf("%d\n", v[id]);
        v[id] = 1;//255-v[id];
    }
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = 16;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}


