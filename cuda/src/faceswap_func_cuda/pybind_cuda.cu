#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include "pybind_cuda.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void gpu_square(int* v, int size){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ){
        v[id] = 255-v[id];
    }
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = BLOCK_SIZE_X*BLOCK_SIZE_Y;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}


