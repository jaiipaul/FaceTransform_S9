#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include "pybind_cuda.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void gpu_square(int* v, int size){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if ( id < size ){
        v[id] = (255-v[id]);
    }
}

__global__ void kernel_ApplyAllHomography(int width, int height, int* imgLabel, double** H, double* XI, double* YI){
    int i = blockIdx.x*blockDim.x + threadIdx.x,
		j = blockIdx.y*blockDim.y + threadIdx.y;
    double *h, den;
    if (i < width  && j < height){
        if (imgLabel[j*width+i] > 0){
            h = H[imgLabel[j*width+i]];

            den = 1.0f / (h[2]*i+h[5]*j+h[8]);
            XI[j*width + i] =(h[0]*i+h[3]*j+h[6])*den;
            YI[j*width + i] =(h[1]*i+h[4]*j+h[7])*den;
        }
    }
    __syncthreads();
}   

__global__ void kernel_RecreateImage(int* imgOut, 
                              int* imgCAM ,int width_CAM, int height_CAM, 
                              int* imgFTA, int width_FTA, int height_FTA, 
                              double* XI, double* YI, int* imgLabel){
    int x_CAM = blockIdx.x*blockDim.x + threadIdx.x,
		y_CAM = blockIdx.y*blockDim.y + threadIdx.y;

    int canal;
    int x_FTA, y_FTA;

    if (x_CAM < width_CAM  && y_CAM < height_CAM){
        if(imgLabel[y_CAM*width_CAM+x_CAM] > 0){
            x_FTA = (int)floor(XI[y_CAM * width_CAM + x_CAM]);
            y_FTA = (int)floor(YI[y_CAM * width_CAM + x_CAM]);  

            for(canal = 0; canal < 3; canal++){
                imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + canal] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + canal];
            }         
        }else{
            for(canal = 0; canal < 3; canal++){
                imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + canal] = imgCAM[y_CAM * 3*width_CAM + x_CAM*3 + canal];
            }
        }
    }    
    __syncthreads();       
}


void gpu_square_main(int * v_in, int size, int * v_out)
{
    int nbThread = BLOCK_SIZE_X*BLOCK_SIZE_Y;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    int * v_in_cuda = NULL;
    hipMalloc((void **)&v_in_cuda, size*sizeof(int));
    hipMemcpy(v_in_cuda, v_in, size*sizeof(int), hipMemcpyHostToDevice);
    gpu_square<<<dimGrid, dimBlock>>>(v_in_cuda, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    //printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(v_out, v_in_cuda, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(v_in_cuda);
}

void ApplyAllHomography(int width, int height, int* imgLabel, double** H, double* XI, double* YI, int n_quadrangles)
{   
    int size = width*height;
    int nbThread = BLOCK_SIZE_X*BLOCK_SIZE_Y;
    dim3 dimBlock(nbThread);
    dim3 dimGrid((size+nbThread-1)/nbThread);
    double * XI_out_cuda = NULL;
    double * YI_out_cuda = NULL;

    /* Copy Homography matrix */
    double ** H_in_cuda = NULL;
    hipMalloc((void **)&H_in_cuda, n_quadrangles*sizeof(double*));
    hipMalloc((void **)&H_in_cuda[0], 9*n_quadrangles*sizeof(double));
    for(int i = 0; i < n_quadrangles; i++){
        H_in_cuda[i] = H_in_cuda[0] + i*9;
    }
    hipMemcpy(H_in_cuda, H, n_quadrangles*sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(H_in_cuda[0], H[0], 9*n_quadrangles*sizeof(double), hipMemcpyHostToDevice);

    /* Copy Label Image */
    int * ImgLabel_in_cuda = NULL;
    hipMalloc((void **)&ImgLabel_in_cuda, size*sizeof(int));
    hipMemcpy(ImgLabel_in_cuda, imgLabel, size*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&XI_out_cuda, size*sizeof(double));
    hipMalloc((void **)&YI_out_cuda, size*sizeof(double));

    
    kernel_ApplyAllHomography<<<dimGrid, dimBlock>>>(width, height, ImgLabel_in_cuda, H_in_cuda, XI_out_cuda, YI_out_cuda);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    //printf("Kernel DONE\n");
    hipDeviceSynchronize();
    hipMemcpy(XI, XI_out_cuda, size*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(XI, XI_out_cuda, size*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(ImgLabel_in_cuda);
    hipFree(H_in_cuda[0]);
    hipFree(H_in_cuda);
    hipFree(XI_out_cuda);
    hipFree(YI_out_cuda);
}

