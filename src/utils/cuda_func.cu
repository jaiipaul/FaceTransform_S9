#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "cuda_func.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void kernel_ApplyAllHomography(int width, int height, 
                                          int* imgLabel, float* H_flat, 
                                          float* XI,    float* YI){
    int i = blockIdx.x*blockDim.x + threadIdx.x,
		j = blockIdx.y*blockDim.y + threadIdx.y;
    float den;
    int label;
    if (i < width  && j < height){
        //printf("%d | %d\n", i, j);
        label = imgLabel[j*width+i];
        if (label > 0){
            label -= 1;
            den = 1.0 / (H_flat[9*label + 2]*i + H_flat[9*label + 5]*j + 1);
            
            XI[j*width + i] = (H_flat[9*label    ]*i+H_flat[9*label + 3]*j+H_flat[9*label + 6])*den;
            YI[j*width + i] = (H_flat[9*label + 1]*i+H_flat[9*label + 4]*j+H_flat[9*label + 7])*den;
        }
    }
}   

__global__ void kernel_RecreateImage(int* imgOut, 
                                     int* imgCAM ,int width_CAM, int height_CAM, 
                                     int* imgFTA, int width_FTA, int height_FTA, 
                                     float* XI, float* YI, int* imgLabel){
    int x_CAM = blockIdx.x*blockDim.x + threadIdx.x,
		y_CAM = blockIdx.y*blockDim.y + threadIdx.y;

    int x_FTA, y_FTA;

    if (x_CAM < width_CAM  && y_CAM < height_CAM){
        if(imgLabel[y_CAM*width_CAM+x_CAM] > 0){
            x_FTA = (int)XI[y_CAM * width_CAM + x_CAM];
            y_FTA = (int)YI[y_CAM * width_CAM + x_CAM];  
  
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 0] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 0];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 1] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 1];
            imgOut[y_CAM * 3*width_CAM + x_CAM * 3 + 2] = imgFTA[y_FTA * 3*width_FTA + x_FTA*3 + 2];          
        }
    }          
}

float ApplyAllHomography_CUDA(int width, int height, int* imgLabel, float* H_flat, float* XI, float* YI, int n_quadrangles)
{   
    int size = width*height;
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);
    dim3 dimGrid((width+nbThreadx-1)/nbThreadx, (height+nbThready-1)/nbThready);

    hipEvent_t start, stop;
    float dt = 0.0f;

    /* Copy Homography matrix */
    float *H_in_cuda = NULL;
    hipMalloc((void **)&H_in_cuda, 9*n_quadrangles*sizeof(float));
    hipMemcpy(H_in_cuda, H_flat, 9*n_quadrangles*sizeof(float), hipMemcpyHostToDevice);

    /* Copy Label Image */
    int * ImgLabel_in_cuda = NULL;
    hipMalloc((void **)&ImgLabel_in_cuda, size*sizeof(int));
    hipMemcpy(ImgLabel_in_cuda, imgLabel, size*sizeof(int), hipMemcpyHostToDevice);

    /* Allocate memory for interpolation matrix */
    float *XI_out_cuda = NULL;
    float *YI_out_cuda = NULL;
    hipMalloc((void **)&XI_out_cuda, size*sizeof(float));
    hipMalloc((void **)&YI_out_cuda, size*sizeof(float));
    
    hipEventCreate(&start);
	hipEventCreate(&stop);	
    hipEventRecord(start, 0);
    kernel_ApplyAllHomography<<<dimGrid, dimBlock>>>(width, height, ImgLabel_in_cuda, H_in_cuda, XI_out_cuda, YI_out_cuda);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&dt, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    
    /* Copy device result to host memory */
    hipMemcpy(XI, XI_out_cuda, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(YI, YI_out_cuda, size*sizeof(float), hipMemcpyDeviceToHost);

    /* Free device memory */
    hipFree(ImgLabel_in_cuda);
    hipFree(H_in_cuda);
    hipFree(XI_out_cuda);
    hipFree(YI_out_cuda);

    //std::cout << "Time spent in kernel : " << dt << std::endl;
    return dt;
}

void RecreateImage_CUDA(int* imgOut,
	               int* imgCAM, int width_CAM, int height_CAM,
	               int* imgFTA, int width_FTA, int height_FTA,
 	               float* XI, float* YI, int* imgLabel)
{
	int size_CAM = width_CAM * height_CAM;
    int size_FTA = width_FTA * height_FTA;

	int* imgout_cuda = NULL, *imgCAM_cuda = NULL, *imgFTA_cuda = NULL, *imgLabel_cuda = NULL;
	float *XI_cuda = NULL, *YI_cuda = NULL;
	
    int nbThreadx = BLOCK_SIZE_X, nbThready = BLOCK_SIZE_Y;
    dim3 dimBlock(nbThreadx, nbThready);
    dim3 dimGrid((width_CAM+nbThreadx-1)/nbThreadx, (height_CAM+nbThready-1)/nbThready);

	hipMalloc((void **)&imgCAM_cuda, 3*size_CAM*sizeof(int));
	hipMemcpy(imgCAM_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&imgFTA_cuda, 3*size_FTA*sizeof(int));	
	hipMemcpy(imgFTA_cuda, imgFTA, 3*size_FTA*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&imgout_cuda, 3*size_CAM*sizeof(int));
    hipMemcpy(imgout_cuda, imgCAM, 3*size_CAM*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&XI_cuda, size_CAM * sizeof(float));
	hipMemcpy(XI_cuda, XI, size_CAM * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&YI_cuda, size_CAM * sizeof(float));
	hipMemcpy(YI_cuda, YI, size_CAM * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&imgLabel_cuda, size_CAM * sizeof(int));
	hipMemcpy(imgLabel_cuda, imgLabel, size_CAM * sizeof(int), hipMemcpyHostToDevice);



    /* Kernel */
	kernel_RecreateImage<<<dimGrid, dimBlock>>>(imgout_cuda, 
                                                imgCAM_cuda, width_CAM, height_CAM, 
                                                imgFTA_cuda, width_FTA, height_FTA,
                                                XI_cuda, YI_cuda, imgLabel_cuda);	

	hipDeviceSynchronize();	
	
    /* Copy result to host memory */
    hipMemcpy(imgOut, imgout_cuda, 3*size_CAM*sizeof(int), hipMemcpyDeviceToHost);
	
    /* Free device memory */
    hipFree(XI_cuda);
	hipFree(YI_cuda);
	hipFree(imgCAM_cuda);
	hipFree(imgFTA_cuda);
	hipFree(imgLabel_cuda);
	hipFree(imgout_cuda);	
}